
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <stdint.h>

__device__ uint8_t merge_colors(uint8_t a, uint8_t b, uint8_t c){
    return (a+b+c)/3;
}

__device__ float blur_effect(size_t x, size_t y) {
    float xp = 1920/2;
    float yp = 1080/2;

    float v = ((x-xp)*(x-xp) + (y-yp)*(y-yp)) / (800*800);
    v = 1 - 1 /(1 + v);
    return v > 1 ? 1 : v;
}

__global__ void process_color(size_t width, size_t height,
        uint8_t *a, uint8_t *b, uint8_t *c, uint8_t *res) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx > (width * 3 * height)) return;

    int x = idx % (width*3);
    int y = idx / (width*3);
    int radius = 40 * blur_effect(x/3, y);

    int sum = 0;
    int count = 0;
    for (int i = x - radius*3; i <= x + radius*3; i += 3) {
        for (int j = y - radius; j <= y + radius; j += 1) {
            if ((i < 0) || (i >= width*3) || (j < 0) || (j >= height)) {
                continue;
            }

            if((i-x)*(i-x)/9 + (j-y)*(j-y) > radius*radius){
                continue;
            }

            int l_idx = i + j * width * 3;
            sum += merge_colors(a[l_idx], b[l_idx], c[l_idx]);
            ++count;
        }
    }

    bool is_red = idx % 3 == 0;
    bool is_green = idx % 3 == 1;
    bool is_blue = idx % 3 == 2;

    float be = 1.f;
    res[idx] = (uint8_t)((sum / count) * be + b[idx] * (1 - be));
}

void read_image(const char *path, uint8_t *data,
        size_t start, size_t size) {
    FILE *fp = fopen(path, "r");
    if (fp == NULL) {
        perror("Error while opening the file.\n");
        exit(EXIT_FAILURE);
    }
    fseek(fp, start, SEEK_SET);
    for (int i = 0; i < size; ++i) {
        data[i] = getc(fp);
    }
    fclose(fp);
}

void write_image(const char *path, uint8_t *data,
        size_t start, size_t size) {
    FILE *fp = fopen(path, "r+");
    if (fp == NULL) {
        perror("Error while opening the file.\n");
        exit(EXIT_FAILURE);
    }
    if (fseek(fp, start, SEEK_SET) != 0) {
        perror("Error while seeking.\n");
        exit(EXIT_FAILURE);
    }
    for (int i = 0; i < size; ++i) {
        putc(data[i], fp);
    }
    fclose(fp);
}

void copy_header(const char *from, const char *to, size_t till) {
    FILE *fp_from = fopen(from, "r");
    FILE *fp_to = fopen(to, "w");
    if (fp_from == NULL || fp_to == NULL) {
        perror("Error while opening the file.\n");
        exit(EXIT_FAILURE);
    }
    if (fseek(fp_to, 0, SEEK_SET) != 0) {
        perror("Error while seeking.\n");
        exit(EXIT_FAILURE);
    }
    for (int i = 0; i < till; ++i) {
        putc(getc(fp_from), fp_to);
    }
    fclose(fp_from);
    fclose(fp_to);
}

int main(void) {
    srand(time(NULL));
    printf("start\n");

    int count;
    int err;
    if ((err = hipGetDeviceCount(&count)) != hipSuccess) {
        printf("error: %d\n", err);
        exit(1);
    }
    printf("count: %d\n", count);

    for (int i = 0; i < count; ++i) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("%d maxTexture1D: %d\n", i, prop.maxTexture1D);
        printf("%d maxTexture2D: %d\n", i, prop.maxTexture2D);
        printf("%d maxTexture3D: %d\n", i, prop.maxTexture3D);
        printf("%d name: %s\n", i, prop.name);
    }

    size_t image_size = sizeof(uint8_t) * 3 * 1920 * 1080;
    uint8_t *a_img = (uint8_t *) malloc(image_size);
    uint8_t *b_img = (uint8_t *) malloc(image_size);
    uint8_t *c_img = (uint8_t *) malloc(image_size);
    uint8_t *result_img = (uint8_t *) malloc(image_size);

    uint8_t *d_a, *d_b, *d_c, *d_result;
    hipMalloc((void **) &d_a, image_size);
    hipMalloc((void **) &d_b, image_size);
    hipMalloc((void **) &d_c, image_size);
    hipMalloc((void **) &d_result, image_size);

    size_t start = 0x7a;
    read_image("input/a.bmp", a_img, start, image_size);
    read_image("input/b.bmp", b_img, start, image_size);
    read_image("input/c.bmp", c_img, start, image_size);
    read_image("input/a.bmp", result_img, start, image_size);

    hipMemcpy(d_a, a_img, image_size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b_img, image_size, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c_img, image_size, hipMemcpyHostToDevice);

    printf("STARTED\n");
    clock_t t;
    t = clock();
    process_color <<< 1920 * 1080 * 3 / 512 + 1, 512 >>>
        (1920, 1080, d_a, d_b, d_c, d_result);
    hipDeviceSynchronize();
    t = clock() - t;
    double time_taken = ((double)t)/CLOCKS_PER_SEC;
    printf("ENDED\n");
    printf("time: %lf pic/s\n", 1000.f / time_taken);

    hipMemcpy(result_img, d_result, image_size,
        hipMemcpyDeviceToHost);

    copy_header("input/a.bmp", "input/result.bmp", start);
    write_image("input/result.bmp", result_img, start, image_size);

    free(a_img);
    free(b_img);
    free(c_img);
    free(result_img);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_result);
    return 0;
}
